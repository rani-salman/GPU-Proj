#include "hip/hip_runtime.h"
#include <assert.h>
#include "common.h"
#include "timer.h"

__global__ void kernel_nw0(unsigned char* sequence1, unsigned char* sequence2, int* scores_d, unsigned int numSequences,
                           int* curr, int* ref_b, int* ref_hv)
{
    int Iteration = 1;

    for (unsigned int i = 0; i <= (2 * SEQUENCE_LENGTH) - 1; ++i) {
        int column = threadIdx.x + 1;
        int row = Iteration;

        if (column <= i + 1 && row <= SEQUENCE_LENGTH && column <= SEQUENCE_LENGTH) {
            ++Iteration;
            int top = (row == 1) ? (column) * DELETION : (ref_b[column - 1]); 
            int left = (column == 1) ? (row) * INSERTION : (ref_b[column - 2]); 
            int topleft = (row == 1) ? (column - 1) * DELETION : (column == 1) ? (row - 1) * INSERTION : (ref_hv[column - 2]);
            int insertion = top + INSERTION;
            int deletion = left + DELETION;
            int match = topleft + ((sequence2[blockIdx.x * SEQUENCE_LENGTH + (row - 1)] == sequence1[blockIdx.x * SEQUENCE_LENGTH + (column - 1)]) ? MATCH : MISMATCH); 
            int max = (insertion > deletion) ? insertion : deletion;
            max = (match > max) ? match : max;
            curr[column - 1] = max;
        }

        __syncthreads();
        if (column <= min(SEQUENCE_LENGTH, i + 1)) {
                if (column <= min(SEQUENCE_LENGTH, i + 1)) {
                    ref_hv[column - 1] = ref_b[column - 1];
                    ref_b[column - 1] = curr[column - 1];
                }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        scores_d[blockIdx.x] = curr[SEQUENCE_LENGTH - 1];
    }
}

void nw_gpu0(unsigned char* sequence1_d, unsigned char* sequence2_d, int* scores_d, unsigned int numSequences) {
    int *curr_d, *ref_b_d, *ref_hv_d;
    hipMalloc((void**)&curr_d, SEQUENCE_LENGTH * sizeof(int));
    hipMalloc((void**)&ref_b_d, SEQUENCE_LENGTH * sizeof(int));
    hipMalloc((void**)&ref_hv_d, SEQUENCE_LENGTH * sizeof(int));

    const unsigned int numThreadsPerBlock = SEQUENCE_LENGTH;
    const unsigned int numBlocks = numSequences;
    kernel_nw0<<<numBlocks, numThreadsPerBlock>>>(sequence1_d, sequence2_d, scores_d, numSequences,
                                                   curr_d, ref_b_d, ref_hv_d);
    hipFree(curr_d);
    hipFree(ref_b_d);
    hipFree(ref_hv_d);
}


